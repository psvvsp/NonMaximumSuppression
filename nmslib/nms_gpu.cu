#include "hip/hip_runtime.h"
#include "NMS_gpu.h"

#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <cassert>
#include <chrono>
#include <iostream>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#ifndef max
#define max(x, y) (((x) > (y))?(x):(y))
#endif

#ifndef min
#define min(x, y) (((x) < (y))?(x):(y))
#endif

__device__
inline integer area(const Box& b)
{
    return (b.right - b.left + 1) * (b.bottom - b.top + 1);
}

__device__
inline real iou_device(const Box& b1, const Box& b2)
{
    Box intersection;

    intersection.left = max(b1.left, b2.left);
    intersection.right = min(b1.right, b2.right);

    if (intersection.left > intersection.right)
        return real(0.0);

    intersection.top = max(b1.top, b2.top);
    intersection.bottom = min(b1.bottom, b2.bottom);

    if (intersection.top > intersection.bottom)
        return real(0.0);

    integer intersection_area = area(intersection);
    integer union_area = area(b1) + area(b2) - intersection_area;

    return real(intersection_area) / real(union_area);
}

//__global__ void NmsKernel(const Box *boxes, real threshold, char *result)
//{
//    extern __shared__ Box boxesShared[];
//    char *resultShared = (char *)(boxesShared + blockDim.x);
//
//    unsigned j = threadIdx.x;
//    boxesShared[j] = boxes[j];
//    resultShared[j] = 0;
//
//    __syncthreads();
//
//    for (unsigned i = 0; i < blockDim.x; i++) {
//        if (resultShared[i]) continue;
//        if (j > i && resultShared[j] == 0 && iou_device(boxesShared[i], boxesShared[j]) >= threshold) {
//            resultShared[j] = 1;
//        }
//        __syncthreads();
//    }
//
//    result[j] = resultShared[j];
//}

__global__ void NmsTriangle(const Box *boxes, real threshold, char *result, unsigned i0, unsigned size)
{
    unsigned j = i0 + threadIdx.x;
    if (j >= size) return;
 
    extern __shared__ Box boxesShared[];
    char *resultShared = (char *)(boxesShared + blockDim.x);

    boxesShared[threadIdx.x] = boxes[j];
    resultShared[threadIdx.x] = result[j];

    __syncthreads();

    for (unsigned i = 0; i < min(blockDim.x, size); i++) {
        if (resultShared[i]) continue;
        if (threadIdx.x > i && resultShared[threadIdx.x] == 0 && iou_device(boxesShared[i], boxesShared[threadIdx.x]) >= threshold) {
            resultShared[threadIdx.x] = 1;
        }
        __syncthreads();
    }

    result[j] = resultShared[threadIdx.x];
}

__global__ void NmsRectangle(const Box* boxes, real threshold, char* result, unsigned i0, unsigned size)
{
    unsigned i = i0 + threadIdx.x;
    if (i >= size) return;
    
    Box box = boxes[i];
    bool res = false;

    unsigned j0 = blockIdx.x * blockDim.x;

    for (unsigned j = j0; j < min(j0 + blockDim.x, size); j++) {
        if (result[j]) continue;
        if (iou_device(box, boxes[j]) >= threshold)
            res = true;
    }

    if (res)
        result[i] = 1;
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

bool nms_gpu(
    const std::vector<Box>& boxesIn,
    const std::vector<real>& scoresIn,
    real threshold,
    std::vector<Box>& boxesOut,
    std::vector<real>& scoresOut)
{
    Box* boxesInBuffer = nullptr;
    real* scoresInBuffer = nullptr;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&boxesInBuffer, boxesIn.size() * sizeof(Box));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&scoresInBuffer, scoresIn.size() * sizeof(real));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(boxesInBuffer, boxesIn.data(), boxesIn.size() * sizeof(Box), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(scoresInBuffer, scoresIn.data(), scoresIn.size() * sizeof(real), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //// Launch a kernel on the GPU with one thread for each element.
    //addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}

    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    // Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

Error:
    hipFree(boxesInBuffer);
    hipFree(scoresInBuffer);

    return cudaStatus;
}

NMS_gpu::NMS_gpu(size_t boxesCountMax)
    : m_boxesInCPU(nullptr)
    , m_boxesInGPU(nullptr)
    , m_scoresInGPU(nullptr)
    , m_resultGPU(nullptr)
    , m_resultCPU(nullptr)
    , m_boxesCountMax(boxesCountMax)
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    cudaStatus = hipHostMalloc(
        (void**)&m_boxesInCPU, m_boxesCountMax * sizeof(Box));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipHostMalloc failed!");
    }

    cudaStatus = hipMalloc(
        (void**)&m_boxesInGPU, m_boxesCountMax * sizeof(Box));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc(
        (void**)&m_scoresInGPU, m_boxesCountMax * sizeof(real));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc(
        (void**)&m_resultGPU, m_boxesCountMax * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipHostMalloc(
        (void**)&m_resultCPU, m_boxesCountMax * sizeof(char));

}

NMS_gpu::~NMS_gpu()
{
    hipHostFree(m_resultCPU);
    hipHostFree(m_boxesInCPU);
    hipFree(m_boxesInGPU);
    hipFree(m_scoresInGPU);
    hipFree(m_resultGPU);
}

namespace {

    struct Record
    {
        Box box;
        real score;
    };
}

bool NMS_gpu::doIt(
    const std::vector<Box>& boxesIn,
    const std::vector<real>& scoresIn,
    real threshold,
    std::vector<Box>& boxesOut,
    std::vector<real>& scoresOut)
{
    assert(boxesIn.size() == scoresIn.size());
    const size_t size = boxesIn.size();

    if (size > m_boxesCountMax)
        return false;

    std::vector<Record> records;
    records.reserve(size);
    for (size_t i = 0; i < size; i++)
        records.push_back({ boxesIn[i], scoresIn[i] });

    using namespace std::chrono;

    // cpu
    steady_clock::time_point t1 = steady_clock::now();

    std::sort(records.begin(), records.end(),
        [](const Record& l, const Record& r) { return l.score > r.score; });

    steady_clock::time_point t2 = steady_clock::now();

    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);

    std::cout << "Sorting took " << time_span.count() << " seconds" << std::endl;

    for (size_t i = 0; i < size; i++)
        m_boxesInCPU[i] = records[i].box;

    hipError_t cudaStatus = hipMemcpy(
        m_boxesInGPU, m_boxesInCPU,
        size * sizeof(Box), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return false;
    }

    t1 = steady_clock::now();

    const size_t BLOCK_SIZE = 1024;
    const size_t numLaunches = (size / BLOCK_SIZE + ((size % BLOCK_SIZE) ? 1 : 0));
    for (size_t i = 0; i < numLaunches; i++) {
        
        if (i > 0)
            NmsRectangle << <i, BLOCK_SIZE >> > (
                m_boxesInGPU, threshold, m_resultGPU, static_cast<unsigned>(i * BLOCK_SIZE), static_cast<unsigned>(size));
        
        NmsTriangle<<<1, BLOCK_SIZE, (sizeof(Box) + 1)* BLOCK_SIZE >> > (
            m_boxesInGPU, threshold, m_resultGPU, static_cast<unsigned>(i * BLOCK_SIZE), static_cast<unsigned>(size));
    }

    //NmsKernel << <1, static_cast<unsigned>(size), (sizeof(Box) + 1)*size >> > (m_boxesInGPU, threshold, m_resultGPU);

    ////// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return false;
    }

    ////// hipDeviceSynchronize waits for the kernel to finish, and returns
    ////// any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        fprintf(stderr, "error string: %s\n", hipGetErrorString(cudaStatus));
        return false;
    }

    t2 = steady_clock::now();

    time_span = duration_cast<duration<double>>(t2 - t1);

    std::cout << "Kernel took " << time_span.count() << " seconds" << std::endl;

    cudaStatus = hipMemcpy(
        m_resultCPU, m_resultGPU,
        size * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return false;
    }

    boxesOut.clear();
    scoresOut.clear();

    for (size_t i = 0; i < size; i++) {
        if (!m_resultCPU[i]) {
            boxesOut.push_back(records[i].box);
            scoresOut.push_back(records[i].score);
        }
    }

    return true;
}
