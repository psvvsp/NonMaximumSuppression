#include "hip/hip_runtime.h"
#include "NMS_gpu.h"

#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <cassert>
#include <chrono>
#include <iostream>

#ifndef max
#define max(x, y) (((x) > (y))?(x):(y))
#endif

#ifndef min
#define min(x, y) (((x) < (y))?(x):(y))
#endif

__device__
inline integer area(const Box& b)
{
    return (b.right - b.left + 1) * (b.bottom - b.top + 1);
}

__device__
inline real iou_device(const Box& b1, const Box& b2)
{
    Box intersection;

    intersection.left = max(b1.left, b2.left);
    intersection.right = min(b1.right, b2.right);

    if (intersection.left > intersection.right)
        return real(0.0);

    intersection.top = max(b1.top, b2.top);
    intersection.bottom = min(b1.bottom, b2.bottom);

    if (intersection.top > intersection.bottom)
        return real(0.0);

    integer intersection_area = area(intersection);
    integer union_area = area(b1) + area(b2) - intersection_area;

    return real(intersection_area) / real(union_area);
}

__global__ void NmsTriangle(const Box *boxes, real threshold, char *result, unsigned i0, unsigned size)
{
    unsigned j = i0 + threadIdx.x;
    if (j >= size) return;
 
    extern __shared__ Box boxesShared[];
    char *resultShared = (char *)(boxesShared + blockDim.x);

    boxesShared[threadIdx.x] = boxes[j];
    resultShared[threadIdx.x] = result[j];

    __syncthreads();

    for (unsigned i = 0; i < min(blockDim.x, size); i++) {
        if (resultShared[i]) continue;
        if (threadIdx.x > i && resultShared[threadIdx.x] == 0 && iou_device(boxesShared[i], boxesShared[threadIdx.x]) >= threshold) {
            resultShared[threadIdx.x] = 1;
        }
        __syncthreads();
    }

    result[j] = resultShared[threadIdx.x];
}

__global__ void NmsRectangle(const Box* boxes, real threshold, char* result, unsigned i0, unsigned size)
{
    unsigned i = i0 + threadIdx.x;
    if (i >= size) return;
    
    Box box = boxes[i];
    bool res = false;

    unsigned j0 = blockIdx.x * blockDim.x;

    for (unsigned j = j0; j < min(j0 + blockDim.x, size); j++) {
        if (result[j]) continue;
        if (iou_device(box, boxes[j]) >= threshold)
            res = true;
    }

    if (res)
        result[i] = 1;
}

NMS_gpu::NMS_gpu()
    : m_boxesCountMax(0)
    , m_maxBlockSize(0)
    , m_boxesInCPU(nullptr)
    , m_boxesInGPU(nullptr)
    , m_resultCPU(nullptr)
    , m_resultGPU(nullptr)
{
}

bool NMS_gpu::init(size_t boxesCountMax)
{
    assert(m_boxesCountMax == 0);
    m_boxesCountMax = boxesCountMax;

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) return false;

    hipDeviceProp_t* props = new hipDeviceProp_t();
    cudaStatus = hipGetDeviceProperties(props, 0);
    m_maxBlockSize = props->maxThreadsPerBlock;
    delete props;
    if (cudaStatus != hipSuccess) return false;

    cudaStatus = hipHostMalloc(
        (void**)&m_boxesInCPU, m_boxesCountMax * sizeof(Box));
    if (cudaStatus != hipSuccess) return false;

    cudaStatus = hipMalloc(
        (void**)&m_boxesInGPU, m_boxesCountMax * sizeof(Box));
    if (cudaStatus != hipSuccess) return false;

    cudaStatus = hipHostMalloc(
        (void**)&m_resultCPU, m_boxesCountMax * sizeof(char));
    if (cudaStatus != hipSuccess) return false;

    cudaStatus = hipMalloc(
        (void**)&m_resultGPU, m_boxesCountMax * sizeof(char));
    if (cudaStatus != hipSuccess) return false;

    return true;
}

NMS_gpu::~NMS_gpu()
{
    hipHostFree(m_resultCPU);
    hipHostFree(m_boxesInCPU);
    hipFree(m_boxesInGPU);
    hipFree(m_resultGPU);
}

namespace {

    struct Record
    {
        Box box;
        real score;
    };
}

bool NMS_gpu::doIt(
    const std::vector<Box>& boxesIn,
    const std::vector<real>& scoresIn,
    real threshold,
    std::vector<Box>& boxesOut,
    std::vector<real>& scoresOut)
{
    assert(boxesIn.size() == scoresIn.size());
    const size_t size = boxesIn.size();

    if (size > m_boxesCountMax)
        return false;

    std::vector<Record> records;
    records.reserve(size);
    for (size_t i = 0; i < size; i++)
        records.push_back({ boxesIn[i], scoresIn[i] });

    //using namespace std::chrono;

    //steady_clock::time_point t1 = steady_clock::now();

    std::sort(records.begin(), records.end(),
        [](const Record& l, const Record& r) { return l.score > r.score; });

    //steady_clock::time_point t2 = steady_clock::now();

    //duration<double> time_span = duration_cast<duration<double>>(t2 - t1);

    //std::cout << "Sorting took " << time_span.count() << " seconds" << std::endl;

    for (size_t i = 0; i < size; i++)
        m_boxesInCPU[i] = records[i].box;

    hipError_t cudaStatus = hipMemcpy(
        m_boxesInGPU, m_boxesInCPU,
        size * sizeof(Box), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        return false;
    }

    cudaStatus = hipMemset(m_resultGPU, 0, size);
    if (cudaStatus != hipSuccess) {
        return false;
    }

    //t1 = steady_clock::now();

    const unsigned maxBlockSize = m_maxBlockSize;
    const unsigned numLaunches = static_cast<unsigned>(
        size / maxBlockSize + ((size % maxBlockSize) ? 1 : 0));
    
    for (unsigned i = 0; i < numLaunches; i++) {
        
        if (i > 0)
            NmsRectangle << <i, maxBlockSize >> > (
                m_boxesInGPU, threshold, m_resultGPU,
                i * maxBlockSize, static_cast<unsigned>(size));
        
        NmsTriangle<<<1, maxBlockSize, (sizeof(Box) + 1)* maxBlockSize >> > (
            m_boxesInGPU, threshold, m_resultGPU,
            i * maxBlockSize, static_cast<unsigned>(size));
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return false;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        return false;
    }

    //t2 = steady_clock::now();

    //time_span = duration_cast<duration<double>>(t2 - t1);

    //std::cout << "Kernel took " << time_span.count() << " seconds" << std::endl;

    cudaStatus = hipMemcpy(
        m_resultCPU, m_resultGPU,
        size * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        return false;
    }

    boxesOut.clear();
    scoresOut.clear();

    for (size_t i = 0; i < size; i++) {
        if (!m_resultCPU[i]) {
            boxesOut.push_back(records[i].box);
            scoresOut.push_back(records[i].score);
        }
    }

    return true;
}
